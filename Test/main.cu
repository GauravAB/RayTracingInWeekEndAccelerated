#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <float.h>
#include <fstream>

#include <hiprand/hiprand_kernel.h>
#include "vector.h"
#include "ray.h"
#include "hitable_list.h"
#include "sphere.h"
#include "hitable.h"
#include "camera.h"
#include "material.h"




#define checkCudaErrors(val) check_cuda((val),#val, __FILE__ , __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result)
	{

		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
		hipDeviceReset();
		exit(99);
	}
}

__device__ vec3 color(const ray& r, hitable** world, hiprandState* local_rand_state)
{
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0f, 1.0f, 1.0f);

	for (int i = 0; i < 50; i++)
	{
		hit_record rec;

		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec))
		{
			ray scattered;
			vec3 attenuation;
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state))
			{
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else
			{
				return vec3(0.0, 0.0, 0.0);
			}
		}
		else
		{
			vec3 unit_direction = unit_vector(r.direction());
			float  t = 0.5f * (unit_direction.y() + 1.0f);
			vec3 c =  (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
			
			return cur_attenuation * c;
		}

	}
	
	return vec3(0.0, 0.0, 0.0);
}
__global__ void render_init(int max_x, int max_y, hiprandState* rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if ((i >= max_x) || (j >= max_y))
	{
		return;
	}
	else
	{
		int pixel_index = j * max_x + i;
		unsigned int seed = 1234;

		hiprand_init(seed, pixel_index, 0, &rand_state[pixel_index]);

	}

}

__global__ void render(vec3* fb, int max_x, int max_y, int ns, camera **cam, hitable** world, hiprandState *rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if ((i >= max_x) || (j >= max_y)) return;

	int pixel_index = j * max_x + i ;

	hiprandState local_rand_state = rand_state[pixel_index];

	vec3 col(0, 0, 0);
	
	for (int s = 0; s < ns; s++)
	{
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		ray r = (*cam)->get_ray(u, v);

		col += color(r, world, &local_rand_state);
	}

	rand_state[pixel_index] = local_rand_state;
	col /= float(ns);
	col[0] = sqrt(col[0]);
	col[1] = sqrt(col[1]);
	col[2] = sqrt(col[2]);

	
	fb[pixel_index] = col;
}

__global__ void create_world(hitable** d_list, hitable** d_world,camera** d_camera, int nx, int ny)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{	
		d_list[0] = new sphere(vec3(0, 0, -1), 0.5, new lambertian(vec3(0.1, 0.2, 0.5)));
		d_list[1] = new sphere(vec3(0, -100.5, -1), 100, new lambertian(vec3(0.8, 0.8, 0.0)));
		d_list[2] = new sphere(vec3(1, 0, -1), 0.5, new metal(vec3(0.8, 0.6, 0.2), 0.0));
		d_list[3] = new sphere(vec3(-1, 0, -1), 0.5, new dielectric(1.5));
		d_list[4] = new sphere(vec3(-1, 0, -1), -0.45, new dielectric(1.5));

		*d_world = new hitable_list(d_list, 5);
		*d_camera = new camera(vec3(-2,2,1),vec3(0,0,-1),vec3(0,1,0),20.0,float(nx)/float(ny));
	}
}

__global__ void free_world(hitable** d_list, hitable** d_world, camera** d_camera)
{
	for (int i = 0; i < 5; i++)
	{
		delete ((sphere*)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	
	delete* d_world;
	delete* d_camera;
}

int main()
{
	std::ofstream out("image.ppm");

	int nx = 1200;
	int ny = 600;
	int ns = 100;
	int tx = 8;
	int ty = 8;
	
	std::cerr << "Redering a " << nx << "x" << ny << " image with " << ns << "samples per pixel ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n"; 

	int num_pixels = nx * ny;
	size_t fb_size = num_pixels * sizeof(vec3);

	//allocate FB
	vec3* fb;
	checkCudaErrors(hipMallocManaged((void**)& fb, fb_size));
	
	//allocate random state
	hiprandState* d_rand_state;
	checkCudaErrors(hipMalloc((void**)& d_rand_state, num_pixels * sizeof(hiprandState)));
	

	//hitable world
	hitable** d_list;
	checkCudaErrors(hipMalloc((void**)& d_list, 2 * sizeof(hitable*)));
	hitable** d_world;
	checkCudaErrors(hipMalloc((void**)& d_world, sizeof(hitable*)));
	
	//device camera
	camera** d_camera;
	checkCudaErrors(hipMalloc((void**)& d_camera, sizeof(camera*)));

	create_world << < 1, 1 >> > (d_list, d_world, d_camera,nx,ny);
	
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	
	
	clock_t start, stop;
	start = clock();

	//render on the buffer
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	render_init <<< blocks, threads >>> (nx, ny, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	render <<<blocks, threads>>> (fb, nx, ny, ns, d_camera, d_world, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop = clock();

	//time eval
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;

	std::cerr << "took: " << timer_seconds << " seconds.\n";
	
	//image output
	out << "P3\n" << nx << " " << ny << "\n255\n";

	for (int j = ny - 1; j >= 0; j--)
	{
		for (int i = 0; i < nx; i++)
		{
			size_t pixel_index = j * nx + i;
			
			int ir = int(255.99 * fb[pixel_index].r());
			int ig = int(255.99 * fb[pixel_index].g());
			int ib = int(255.99 * fb[pixel_index].b());
			
			out << ir << " " << ig << " " << ib << std::endl;
		}
	}
	

	checkCudaErrors(hipDeviceSynchronize());
	free_world << <1, 1 >> > (d_list, d_world,d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(fb));

	hipDeviceReset();

	out.close();

}
















	